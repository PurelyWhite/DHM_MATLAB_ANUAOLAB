#include "hip/hip_runtime.h"
//This program is written by Munther Gdeisat and Miguel Arevallilo Herra�ez to program the two-dimensional unwrapper
//entitled "Fast two-dimensional phase-unwrapping algorithm based on sorting by 
//reliability following a noncontinuous path"
//by  Miguel Arevallilo Herra�ez, David R. Burton, Michael J. Lalor, and Munther A. Gdeisat
//published in the Applied Optics, Vol. 41, No. 35, pp. 7437, 2002.
//This program is written on 15th August 2007
//The wrapped phase map is floating point data type. Also, the unwrapped phase map is foloating point
//#include <sys/malloc.h>
#include<stdio.h>
#include <stdlib.h>
#include <string.h>
#include "mex.h"   //--This one is required

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
//#include <thrust/sort.h>
//#include <thrust/device_ptr.h>
//#include <stl.h>

__device__ static float PI = 3.141592654;
__device__ static float TWOPI = 6.283185307;

bool DEBUG = true;

/*
struct saxpy_functor
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
    float operator()(const float& x, const float& y) const {
        return a * x + y;
    }
};

void saxpy_fast(float A, thrust::device_ptr<float> X, thrust::device_ptr<float> Y, int N)
{
    // Y <- A * X + Y
    thrust::transform(X, X+N, Y, Y+N, saxpy_functor(A));
}
*/

__global__ void gpu_derivative_vertical(float* WrappedImage, float* DerivativeImage, int width, int height){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float* WIP;
    float* DIP;

    for(int i=index; i<width*height; i+=stride){
        WIP = WrappedImage + i;
        DIP = DerivativeImage + i;

        if(i/height == 0){
            *DIP = *(WIP+width) - *WIP;
        } else if(i + width > width*height){
            *DIP = *WIP - *(WIP-width);
        } else {
            *DIP = (*(WIP+width)-*WIP+*WIP-*(WIP-width))/2;
        }
    }
}

__global__ void gpu_derivative_horizontal(float* WrappedImage, float* DerivativeImage, int width, int height){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float* WIP;
    float* DIP;

    for(int i=index; i<width*height; i+=stride){
        WIP = WrappedImage + i;
        DIP = DerivativeImage + i;

        if(i % width == 0){
            *DIP = *(WIP+1) - *WIP;
        } else if (i % width == width-1){
            *DIP = *WIP - *(WIP-1);
        } else {
            *DIP = (*(WIP+1)-*WIP+*WIP-*(WIP-1))/2;
        }
    }
}

__global__ void add(float* A, float* B, int N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float* a;
    float* b;

    for(int i=index; i<N; i+=stride){
        a = A + i;
        b = B + i;

        *a = *a + *b;
    }
}

__global__ void double_array(float* A, float* B, int width, int height){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float* a;
    float* b;

    for(int i=index; i<(width)*(height); i+=stride){
        int r = i/width;
        int c = i % width;

        b = B + (r*width*4) + c;

        if(i%width*4 < width){
            a = A + i;
            *b = *a;
        } else {
            *b = 0.0f;
        }
    }
}

__global__ void dct(hipfftComplex* fftData, float* dctData, int width, int height){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i=index; i<width*height; i+=stride){
        int r = i/width;
        int c = i % width;

        //float abs_fft =

        //dctData[i] =
    }
}

__global__ void gpu_unwrap(float* WrappedImage, float* UnwrappedImage, int width, int height){

}

//the main function of the unwrapper
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    //Declarations of getting two arrays from Matlab
    //1)input wrapped image  of type float and 2)mask of type unsigned char
    float *WrappedImage = (float *)mxGetData(prhs[0]);
    int image_width = mxGetM(prhs[0]);
    int image_height = mxGetN(prhs[0]);

    //declare a place to store the unwrapped image and return it to Matlab
    const mwSize *dims = mxGetDimensions(prhs[0]);
    plhs[0] = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    float *UnwrappedImage = (float *)mxGetPr(plhs[0]);

    int image_size = image_height * image_width;
    int two_image_size = 2 * image_size;

    int No_of_Edges = (image_width)*(image_height-1) + (image_width-1)*(image_height);

    int blockSize = 1024; //1024;
    int numBlocks = 64; //(image_size + blockSize - 1) / blockSize;

    float* derivative_x, *derivative_xx, *derivative_y, *derivative_yy, *gpuWrappedImage;

    hipMallocManaged(&derivative_x,image_size * sizeof(float));
    hipMallocManaged(&derivative_xx,image_size * sizeof(float));
    hipMallocManaged(&derivative_y,image_size * sizeof(float));
    hipMallocManaged(&derivative_yy,image_size * sizeof(float));
    hipMallocManaged(&gpuWrappedImage,image_size* sizeof(float));

    hipMemcpy(gpuWrappedImage, WrappedImage, image_size*sizeof(float), hipMemcpyHostToDevice);

    gpu_derivative_horizontal<<<numBlocks,blockSize>>>(gpuWrappedImage,derivative_x,image_width,image_height);
    gpu_derivative_horizontal<<<numBlocks,blockSize>>>(gpuWrappedImage,derivative_y,image_width,image_height);
    gpu_derivative_horizontal<<<numBlocks,blockSize>>>(derivative_x,derivative_xx,image_width,image_height);
    gpu_derivative_horizontal<<<numBlocks,blockSize>>>(derivative_y,derivative_yy,image_width,image_height);

    add<<<numBlocks,blockSize>>>(derivative_xx,derivative_yy,image_size);

    float* double_img;
    hipMallocManaged(&double_img, 8*image_size * sizeof(float));

    double_array<<<numBlocks,blockSize>>>(derivative_xx,double_img,image_width,image_height);

    hipfftHandle plan;
    hipfftComplex* fftData;

    hipMalloc((void**)&fftData, sizeof(hipfftComplex)*(image_width/2+1)*image_height);
    if (hipGetLastError() != hipSuccess){
        mexPrintf("Cuda error: Failed to allocate\n");
        return;
    }

    if (hipfftPlan2d(&plan, image_width,image_height,HIPFFT_R2C) != HIPFFT_SUCCESS){
        mexPrintf("CUFFT error: Plan creation failed");
        return;
    }

    /* Use the CUFFT plan to transform the signal in place.*/
    if (hipfftExecR2C(plan, (hipfftReal*)double_img, fftData) != HIPFFT_SUCCESS){
        mexPrintf("CUFFT error: ExecC2C Forward failed");
        return;
    }

    hipMemcpy(UnwrappedImage, gpuWrappedImage, image_size * sizeof(float), hipMemcpyDeviceToHost);

    if (hipDeviceSynchronize() != hipSuccess){
        mexPrintf("Cuda error: Failed to synchronize\n");
        return;
    }

    hipFree(derivative_x);
    hipFree(derivative_xx);
    hipFree(derivative_y);
    hipFree(derivative_yy);
    hipFree(gpuWrappedImage);
    hipFree(double_img);
    hipfftDestroy(plan);

    return;
}
