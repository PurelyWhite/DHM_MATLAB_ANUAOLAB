#include "hip/hip_runtime.h"
//This program is written by Munther Gdeisat and Miguel Arevallilo Herra�ez to program the two-dimensional unwrapper
//entitled "Fast two-dimensional phase-unwrapping algorithm based on sorting by 
//reliability following a noncontinuous path"
//by  Miguel Arevallilo Herra�ez, David R. Burton, Michael J. Lalor, and Munther A. Gdeisat
//published in the Applied Optics, Vol. 41, No. 35, pp. 7437, 2002.
//This program is written on 15th August 2007
//The wrapped phase map is floating point data type. Also, the unwrapped phase map is foloating point
//#include <sys/malloc.h>
#include<stdio.h>
#include <stdlib.h>
#include <string.h>
#include "mex.h"   //--This one is required

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
//#include <stl.h>

__device__ static float PI = 3.141592654;
__device__ static float TWOPI = 6.283185307;

bool DEBUG = true;

//pixel information
struct PIXEL
{
    //int x;					//x coordinate of the pixel
    //int y;					//y coordinate
    int increment;			//No. of 2*pi to add to the pixel to unwrap it
    int number_of_pixels_in_group;	//No. of pixels in the pixel group
    float value;			//value of the pixel
    float reliability;
    int group;				//group No.
    int new_group;
    struct PIXEL *head;		//pointer to the first pixel in the group in the linked list
    struct PIXEL *last;		//pointer to the last pixel in the group
    struct PIXEL *next;		//pointer to the next pixel in the group
};


//the EDGE is the line that connects two pixels.
//if we have S PIXELs, then we have S horizental edges and S vertical edges
struct EDGE
{
    float reliab;			//reliabilty of the edge and it depends on the two pixels
    PIXEL *pointer_1;		//pointer to the first pixel
    PIXEL *pointer_2;		//pointer to the second pixel
    int increment;			//No. of 2*pi to add to one of the pixels to unwrap it with respect to the second

    bool operator < (const EDGE& edge) const
    {
        return (reliab < edge.reliab);
    }
};

//---------------start quicker_sort algorithm --------------------------------
#define swap(x,y) {EDGE t; t=x; x=y; y=t;}
#define order(x,y) if (x.reliab > y.reliab) swap(x,y)
#define o2(x,y) order(x,y)
#define o3(x,y,z) o2(x,y); o2(x,z); o2(y,z)

typedef enum {yes, no} yes_no;

__device__ bool find_pivot(EDGE *left, EDGE *right, float *pivot_ptr)
{
    EDGE a, b, c, *p;

    a = *left;
    b = *(left + (right - left) /2 );
    c = *right;
    o3(a,b,c);

    if (a.reliab < b.reliab)
    {
        *pivot_ptr = b.reliab;
        return true;
    }

    if (b.reliab < c.reliab)
    {
        *pivot_ptr = c.reliab;
        return true;
    }

    for (p = left + 1; p <= right; ++p)
    {
        if (p->reliab != left->reliab)
        {
            *pivot_ptr = (p->reliab < left->reliab) ? left->reliab : p->reliab;
            return true;
        }
        return false;
    }
}

__device__ EDGE *partition(EDGE *left, EDGE *right, float pivot)
{
    while (left <= right)
    {
        while (left->reliab < pivot)
            ++left;
        while (right->reliab >= pivot)
            --right;
        if (left < right)
        {
            swap (*left, *right);
            ++left;
            --right;
        }
    }
    return left;
}

__device__ void gpu_quicker_sort(EDGE *left, EDGE *right)
{
    EDGE *p;
    float pivot;

    if (find_pivot(left, right, &pivot))
    {
        p = partition(left, right, pivot);
        gpu_quicker_sort(left, p - 1);
        gpu_quicker_sort(p, right);
    }
}

__global__ void quicker_sort(EDGE *left, EDGE *right)
{
    EDGE *p;
    float pivot;

    if (find_pivot(left, right, &pivot))
    {
        p = partition(left, right, pivot);
        gpu_quicker_sort(left, p - 1);
        gpu_quicker_sort(p, right);
    }
}

//--------------end quicker_sort algorithm -----------------------------------

//--------------------start initialse pixels ----------------------------------
//initialse pixels. See the explanation of the pixel class above.
//initially every pixel is a group by its self
__global__
void  initialisePIXELs(float *WrappedImage, PIXEL *pixel, int image_width, int image_height, hiprandState *d_rand_state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    PIXEL *pixel_pointer;
    float *wrapped_image_pointer;

    for(int i = index; i < image_width*image_height; i += stride) {
        pixel_pointer = pixel + i;
        wrapped_image_pointer = WrappedImage + i;
        pixel_pointer->increment = 0;
        pixel_pointer->number_of_pixels_in_group = 1;
        pixel_pointer->value = *wrapped_image_pointer;
        pixel_pointer->reliability = 9999999.0 + hiprand_uniform(&d_rand_state[index]);
        pixel_pointer->head = pixel_pointer;
        pixel_pointer->last = pixel_pointer;
        pixel_pointer->next = NULL;
        pixel_pointer->new_group = 0;
        pixel_pointer->group = -1;
    }
}
//-------------------end initialise pixels -----------

//gamma function in the paper
__device__ float wrap(float pixel_value)
{
    float wrapped_pixel_value;
    if (pixel_value > PI)	wrapped_pixel_value = pixel_value - TWOPI;
    else if (pixel_value < -PI)	wrapped_pixel_value = pixel_value + TWOPI;
    else wrapped_pixel_value = pixel_value;

    return wrapped_pixel_value;
}

// pixelL_value is the left pixel,	pixelR_value is the right pixel
__device__ int find_wrap(float pixelL_value, float pixelR_value)
{
    float difference;
    int wrap_value;
    difference = pixelL_value - pixelR_value;

    if (difference > PI){
        wrap_value = -1;
    }
    else if (difference < -PI){
        wrap_value = 1;
    }
    else {
        wrap_value = 0;
    }

    return wrap_value;
}

__global__ void calculate_reliability(float *wrappedImage, PIXEL *pixel, int image_width, int image_height)
{
    int image_width_plus_one = image_width + 1;
    int image_width_minus_one = image_width - 1;
    PIXEL *pixel_pointer = pixel + image_width_plus_one;
    float *WIP = wrappedImage + image_width_plus_one; //WIP is the wrapped image pointer
    float H, V, D1, D2;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for(int i = index; i < (image_height-1)*(image_width); i += stride){
        // Ignore the first or last pixel in each row
        if(index % image_width == 0 || index % image_width == image_width - 1){
            continue;
        }

        pixel_pointer = pixel + image_width_plus_one + index;
        WIP = wrappedImage + image_width_plus_one + index;

        H = wrap(*(WIP - 1) - *WIP) - wrap(*WIP - *(WIP + 1));
        V = wrap(*(WIP - image_width) - *WIP) - wrap(*WIP - *(WIP + image_width));
        D1 = wrap(*(WIP - image_width_plus_one) - *WIP) - wrap(*WIP - *(WIP + image_width_plus_one));
        D2 = wrap(*(WIP - image_width_minus_one) - *WIP) - wrap(*WIP - *(WIP + image_width_minus_one));
        pixel_pointer->reliability = H*H + V*V + D1*D1 + D2*D2;
    }

    /*
    for (int i = 1; i < image_height -1; ++i)
    {
        for (int j = 1; j < image_width - 1; ++j)
        {
            H = wrap(*(WIP - 1) - *WIP) - wrap(*WIP - *(WIP + 1));
            V = wrap(*(WIP - image_width) - *WIP) - wrap(*WIP - *(WIP + image_width));
            D1 = wrap(*(WIP - image_width_plus_one) - *WIP) - wrap(*WIP - *(WIP + image_width_plus_one));
            D2 = wrap(*(WIP - image_width_minus_one) - *WIP) - wrap(*WIP - *(WIP + image_width_minus_one));
            pixel_pointer->reliability = H*H + V*V + D1*D1 + D2*D2;
            pixel_pointer++;
            WIP++;
        }
        pixel_pointer += 2;
        WIP += 2;
    }*/
}

//calculate the reliability of the horizental edges of the image
//it is calculated by adding the reliability of pixel and the relibility of
//its right neighbour
//edge is calculated between a pixel and its next neighbour
__global__ void horizontalEDGEs(PIXEL *pixel, EDGE *edge, int image_width, int image_height)
{
    EDGE *edge_pointer = edge;
    PIXEL *pixel_pointer = pixel;
    char mybuff1[50];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for(int i = index; i < image_height*image_width; i += stride){
        if(i % image_width == image_width - 1){
            continue;
        }
        pixel_pointer = pixel + i;
        int edge_pos = (i % (image_width)) + ((i/(image_width))*(image_width-1));
        edge_pointer = edge + edge_pos;

        edge_pointer->pointer_1 = pixel_pointer;
        edge_pointer->pointer_2 = (pixel_pointer+1);
        edge_pointer->reliab = pixel_pointer->reliability + (pixel_pointer + 1)->reliability;
        edge_pointer->increment = find_wrap(pixel_pointer->value, (pixel_pointer + 1)->value);
    }

    /*
    for (int i = 0; i < image_height; i++)
    {
        for (int j = 0; j < image_width - 1; j++)
        {
            edge_pointer->pointer_1 = pixel_pointer;
            edge_pointer->pointer_2 = (pixel_pointer+1);
            edge_pointer->reliab = pixel_pointer->reliability + (pixel_pointer + 1)->reliability;
            edge_pointer->increment = find_wrap(pixel_pointer->value, (pixel_pointer + 1)->value);
            if(edge_pointer->increment != 0){
                //mexPrintf("Inc\n");
                //sprintf (mybuff1, "%d", edge_pointer->increment);
                //mexPrintf(mybuff1);
                //mexPrintf("\n");
            }
            pixel_pointer++;
            edge_pointer++;
        }
        pixel_pointer++;
    }*/
}

//calculate the reliability of the vertical EDGEs of the image
//it is calculated by adding the reliability of pixel and the relibility of
//its lower neighbour in the image.
__global__ void  verticalEDGEs(PIXEL *pixel, EDGE *edge, int image_width, int image_height)
{
    PIXEL *pixel_pointer = pixel;
    EDGE *edge_pointer = edge + (image_height) * (image_width - 1);
    char mybuff1[50];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < (image_height-1)*(image_width); i += stride) {
        pixel_pointer = pixel + i;
        edge_pointer = edge + i + ((image_height) * (image_width - 1));

        edge_pointer->pointer_1 = pixel_pointer;
        edge_pointer->pointer_2 = (pixel_pointer + image_width);
        edge_pointer->reliab = pixel_pointer->reliability + (pixel_pointer + image_width)->reliability;
        edge_pointer->increment = find_wrap(pixel_pointer->value, (pixel_pointer + image_width)->value);
    }

    /*
    for (int i=0; i<image_height - 1; i++)
    {
        for (int j=0; j < image_width; j++)
        {
            edge_pointer->pointer_1 = pixel_pointer;
            edge_pointer->pointer_2 = (pixel_pointer + image_width);
            edge_pointer->reliab = pixel_pointer->reliability + (pixel_pointer + image_width)->reliability;
            edge_pointer->increment = find_wrap(pixel_pointer->value, (pixel_pointer + image_width)->value);
            if(edge_pointer->increment != 0){
                //mexPrintf("Inc\n");
                //sprintf (mybuff1, "%d", edge_pointer->increment);
                //mexPrintf(mybuff1);
                //mexPrintf("\n");
            }
            pixel_pointer++;
            edge_pointer++;
        } //j loop
    } // i loop
    */
}

//gather the pixels of the image into groups
__global__ void  gatherPIXELs(EDGE *edge, int image_width, int image_height)
{
    int k;
    char mybuff1[50], mybuff2[50],mybuff3[50];

    //Number of rialiable edges (not at the borders of the image)
    int no_EDGEs = (image_width - 1) * (image_height) + (image_width) * (image_height - 1);
    PIXEL *PIXEL1;
    PIXEL *PIXEL2;

    PIXEL *group1;
    PIXEL *group2;
    EDGE *pointer_edge = edge;
    int incremento;

    for (k = 0; k < no_EDGEs; k++)
    {
        PIXEL1 = pointer_edge->pointer_1;
        PIXEL2 = pointer_edge->pointer_2;

        //PIXEL 1 and PIXEL 2 belong to different groups
        //initially each pixel is a group by it self and one pixel can construct a group
        //no else or else if to this if
        if (PIXEL2->head != PIXEL1->head)
        {

            /*sprintf (mybuff1, "%f", pointer_edge->reliab);
            sprintf (mybuff2, "%f", PIXEL1->value);
            sprintf (mybuff3, "%f", PIXEL2->value);
            mexPrintf("Pix A: ");
            mexPrintf(mybuff2);
            mexPrintf(" - Pix B: ");
            mexPrintf(mybuff3);
            mexPrintf(" - ");
            mexPrintf(mybuff1);
            mexPrintf(" - ");*/
            //PIXEL 2 is alone in its group
            //merge this pixel with PIXEL 1 group and find the number of 2 pi to add
            //to or subtract to unwrap it
            if ((PIXEL2->next == NULL) && (PIXEL2->head == PIXEL2))
            {
                //mexPrintf("New B\n");
                PIXEL1->head->last->next = PIXEL2;
                PIXEL1->head->last = PIXEL2;
                (PIXEL1->head->number_of_pixels_in_group)++;
                PIXEL2->head=PIXEL1->head;
                PIXEL2->increment = PIXEL1->increment-pointer_edge->increment;
            }

                //PIXEL 1 is alone in its group
                //merge this pixel with PIXEL 2 group and find the number of 2 pi to add
                //to or subtract to unwrap it
            else if ((PIXEL1->next == NULL) && (PIXEL1->head == PIXEL1))
            {
                //mexPrintf("New A\n");
                PIXEL2->head->last->next = PIXEL1;
                PIXEL2->head->last = PIXEL1;
                (PIXEL2->head->number_of_pixels_in_group)++;
                PIXEL1->head = PIXEL2->head;
                PIXEL1->increment = PIXEL2->increment+pointer_edge->increment;
            }

                //PIXEL 1 and PIXEL 2 both have groups
            else
            {
                group1 = PIXEL1->head;
                group2 = PIXEL2->head;
                //the no. of pixels in PIXEL 1 group is large than the no. of PIXELs
                //in PIXEL 2 group.   Merge PIXEL 2 group to PIXEL 1 group
                //and find the number of wraps between PIXEL 2 group and PIXEL 1 group
                //to unwrap PIXEL 2 group with respect to PIXEL 1 group.
                //the no. of wraps will be added to PIXEL 2 grop in the future
                if (group1->number_of_pixels_in_group > group2->number_of_pixels_in_group)
                {
                    //mexPrintf("Big A\n");
                    //merge PIXEL 2 with PIXEL 1 group
                    group1->last->next = group2;
                    group1->last = group2->last;
                    group1->number_of_pixels_in_group = group1->number_of_pixels_in_group + group2->number_of_pixels_in_group;
                    incremento = PIXEL1->increment-pointer_edge->increment - PIXEL2->increment;
                    //merge the other pixels in PIXEL 2 group to PIXEL 1 group
                    while (group2 != NULL)
                    {
                        group2->head = group1;
                        group2->increment += incremento;
                        group2 = group2->next;
                    }
                }

                    //the no. of PIXELs in PIXEL 2 group is large than the no. of PIXELs
                    //in PIXEL 1 group.   Merge PIXEL 1 group to PIXEL 2 group
                    //and find the number of wraps between PIXEL 2 group and PIXEL 1 group
                    //to unwrap PIXEL 1 group with respect to PIXEL 2 group.
                    //the no. of wraps will be added to PIXEL 1 grop in the future
                else
                {
                    //mexPrintf("Big B\n");
                    //merge PIXEL 1 with PIXEL 2 group
                    group2->last->next = group1;
                    group2->last = group1->last;
                    group2->number_of_pixels_in_group = group2->number_of_pixels_in_group + group1->number_of_pixels_in_group;
                    incremento = PIXEL2->increment + pointer_edge->increment - PIXEL1->increment;
                    //merge the other pixels in PIXEL 2 group to PIXEL 1 group
                    while (group1 != NULL)
                    {
                        group1->head = group2;
                        group1->increment += incremento;
                        group1 = group1->next;
                    } // while
                } // else
            } //else
        } else {
            //mexPrintf("Same group\n");
        };//if

        pointer_edge++;
    }
}

//unwrap the image
__global__ void unwrapImage(PIXEL *pixel, int image_width, int image_height)
{
    int image_size = image_width * image_height;
    PIXEL *pixel_pointer;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < image_size; i += stride) {
        pixel_pointer = pixel + i;
        pixel_pointer->value += TWOPI * (float)(pixel_pointer->increment);
    }
}

//the input to this unwrapper is an array that contains the wrapped phase map.
//copy the image on the buffer passed to this unwrapper to over write the unwrapped
//phase map on the buffer of the wrapped phase map.
__global__ void  returnImage(PIXEL *pixel, float *unwrappedImage, int image_width, int image_height)
{
    int image_size = image_width * image_height;
    float *unwrappedImage_pointer;
    PIXEL *pixel_pointer;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < image_size; i += stride) {
        pixel_pointer = pixel + i;
        unwrappedImage_pointer = unwrappedImage + i;
        *unwrappedImage_pointer = pixel_pointer->value;  //(float) pixel_pointer->reliability;
    }
}

__global__ void init_rand(hiprandState *state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1337, idx,0, state);
}

__global__ void gpuUnwrap(float* WrappedImage, float* UnwrappedImage, int image_width, int image_height, hiprandState *state){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1337, index,0, state);
}

__global__ void init_test_edges(int No_of_Edges, EDGE *test_edges){
    EDGE *test_e;
    for(int i=0; i<No_of_Edges; i++){
        test_e = test_edges + i;

        //if(DEBUG){
        //    mexPrintf("Running test: initialising random value %2d\n", i);
        //}

        test_e->reliab = 0; //rand();
    }
}

//the main function of the unwrapper
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    //Declarations of getting two arrays from Matlab
    //1)input wrapped image  of type float and 2)mask of type unsigned char
    float *WrappedImage = (float *)mxGetData(prhs[0]);
    int image_width = mxGetM(prhs[0]);
    int image_height = mxGetN(prhs[0]);

    //declare a place to store the unwrapped image and return it to Matlab
    const mwSize *dims = mxGetDimensions(prhs[0]);
    plhs[0] = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    float *UnwrappedImage = (float *)mxGetPr(plhs[0]);

    int i, j;
    int image_size = image_height * image_width;
    int two_image_size = 2 * image_size;

    int No_of_Edges = (image_width)*(image_height-1) + (image_width-1)*(image_height);

    int blockSize = 256; //1024;
    int numBlocks = 32; //(image_size + blockSize - 1) / blockSize;

    PIXEL *pixel;
    EDGE *edge;

    if(DEBUG){
        mexPrintf("WrappedImage %2.2f\n", *WrappedImage);
    }

    hipMallocManaged(&pixel,image_size* sizeof(PIXEL));
    hipMallocManaged(&edge,No_of_Edges* sizeof(EDGE));

    float* gpuWrappedImage;

    hipMallocManaged(&gpuWrappedImage,image_size* sizeof(float));
    hipMemcpy(gpuWrappedImage, WrappedImage, image_size*sizeof(float), hipMemcpyHostToDevice);

    //initialise the pixels
    if(DEBUG){
        mexPrintf("Initialising pixels\n");
    }

    //mexPrintf("Initialising Random Number")
    hiprandState *d_rand_state;
    hipMallocManaged(&d_rand_state,blockSize*numBlocks);

    init_rand<<<numBlocks,blockSize>>>(d_rand_state);
    initialisePIXELs<<<numBlocks,blockSize>>>(gpuWrappedImage, pixel, image_width, image_height, d_rand_state);

    //PIXEL *gpu_pixel;
    //EDGE *gpu_edge;

    //hipMalloc(&pixel, image_size*sizeof(PIXEL));
    //hipMalloc(&edge, No_of_Edges*sizeof(EDGE));

    //hipMemcpy(gpu_pixel, pixel, image_size*sizeof(PIXEL), hipMemcpyHostToDevice);
    //hipMemcpy(gpu_edge, edge, No_of_Edges * sizeof(EDGE),hipMemcpyHostToDevice);

    if(DEBUG){
        mexPrintf("Calculating reliabililty\n");
    }

    calculate_reliability<<<numBlocks,blockSize>>>(gpuWrappedImage, pixel, image_width, image_height); //

    //PIXEL *pixel_pointer = pixel;
    //char mybuff1[50],mybuff2[50],mybuff3[50],mybuff4[50];

    if(DEBUG){
        mexPrintf("Gathering edges\n");
    }

    horizontalEDGEs<<<numBlocks,numBlocks>>>(pixel, edge, image_width, image_height);
    verticalEDGEs<<<numBlocks,blockSize>>>(pixel, edge, image_width, image_height);

    if(DEBUG){
        mexPrintf("Sorting edges\n");
    }

    //sort the EDGEs depending on their reliability. The PIXELs with higher reliability (small value) first
    //if your code stuck because of the quicker_sort() function, then use the quick_sort() function
    //run only one of the two functions (quick_sort() or quicker_sort() )
    //quick_sort(edge, No_of_Edges);
    quicker_sort<<<1,1>>>(edge, edge + No_of_Edges - 1);

    if(DEBUG){
        mexPrintf("Running test\n");
    }

    const int N = 6;
    int A[N] = {1, 4, 2, 8, 5, 7};

    EDGE *test_edges;
    hipMallocManaged(&test_edges,No_of_Edges* sizeof(EDGE));

    thrust::device_ptr<EDGE> device_test_edges(test_edges);

    if(DEBUG){
        mexPrintf("Running test: initialising random values\n");
    }

    init_test_edges<<<1,1>>>(No_of_Edges, test_edges);

    if(DEBUG){
        mexPrintf("Running test sort\n");
    }

    //hipDeviceSynchronize();
    //thrust::stable_sort(device_test_edges, device_test_edges + No_of_Edges - 1, thrust::less<EDGE>());

    //thrust::stable_sort(edge,edge+No_of_Edges-1,thrust::less<EDGE>());

    //EDGE *edge_pointer = edge;
    //PIXEL *PIXEL1;
    //PIXEL *PIXEL2;
    //double diff;

    //int a;

    if(DEBUG){
        mexPrintf("Gathering the pixels...\n");
    }

    //gather PIXELs into groups
    gatherPIXELs<<<1,1>>>(edge, image_width, image_height);

    if(DEBUG){
        mexPrintf("Unwrapping Image...\n");
    }

    //unwrap the whole image
    unwrapImage<<<numBlocks,blockSize>>>(pixel, image_width, image_height);

    if(DEBUG){
        mexPrintf("Returning Image...\n");
    }

    float* gpuUnwrappedImage;
    hipMallocManaged(&gpuUnwrappedImage, image_size * sizeof(float));

    //copy the image from PIXEL structure to the wrapped phase array passed to this function
    returnImage<<<numBlocks,blockSize>>>(pixel, gpuUnwrappedImage, image_width, image_height);

    if(DEBUG){
        mexPrintf("Copying unwrapped image...\n");
    }
    hipMemcpy(UnwrappedImage, gpuUnwrappedImage, image_size * sizeof(float), hipMemcpyDeviceToHost);

    if(DEBUG){
        //mexPrintf("Unwrapped Image %2.2f\n", pixel->value);
        //mexPrintf("Unwrapped Image %2.2f\n", *gpuUnwrappedImage);
    }

    hipDeviceSynchronize();

    hipFree(edge);
    hipFree(pixel);
    hipFree(gpuWrappedImage);
    hipFree(gpuUnwrappedImage);
    hipFree(d_rand_state);

    if(DEBUG){
        mexPrintf("Phase successfully retrieved...\n");
    }

    return;
}
