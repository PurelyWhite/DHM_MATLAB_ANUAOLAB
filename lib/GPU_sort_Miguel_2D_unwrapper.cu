//This program is written by Munther Gdeisat and Miguel Arevallilo Herra�ez to program the two-dimensional unwrapper
//entitled "Fast two-dimensional phase-unwrapping algorithm based on sorting by 
//reliability following a noncontinuous path"
//by  Miguel Arevallilo Herra�ez, David R. Burton, Michael J. Lalor, and Munther A. Gdeisat
//published in the Applied Optics, Vol. 41, No. 35, pp. 7437, 2002.
//This program is written on 15th August 2007
//The wrapped phase map is floating point data type. Also, the unwrapped phase map is foloating point
//#include <sys/malloc.h>
#include<stdio.h>
#include <stdlib.h>
#include <string.h>
#include "mex.h"   //--This one is required
#include <thrust/sort.h>

static float PI = 3.141592654;
static float TWOPI = 6.283185307;

//pixel information
struct PIXEL
{
    //int x;					//x coordinate of the pixel
    //int y;					//y coordinate
    int increment;			//No. of 2*pi to add to the pixel to unwrap it
    int number_of_pixels_in_group;	//No. of pixels in the pixel group
    float value;			//value of the pixel
    float reliability;
    int group;				//group No.
    int new_group;
    struct PIXEL *head;		//pointer to the first pixel in the group in the linked list
    struct PIXEL *last;		//pointer to the last pixel in the group
    struct PIXEL *next;		//pointer to the next pixel in the group
};


//the EDGE is the line that connects two pixels.
//if we have S PIXELs, then we have S horizental edges and S vertical edges
struct EDGE
{
    float reliab;			//reliabilty of the edge and it depends on the two pixels
    PIXEL *pointer_1;		//pointer to the first pixel
    PIXEL *pointer_2;		//pointer to the second pixel
    int increment;			//No. of 2*pi to add to one of the pixels to unwrap it with respect to the second

    bool operator < (const EDGE& edge) const
    {
        return (reliab < edge.reliab);
    }
};


//another version of Mixtogether but this function should only be use with the sort program
void  Mix(EDGE *Pointer1, int *index1, int *index2, int size)
{
    int counter1 = 0;
    int counter2 = 0;
    int *TemporalPointer = index1;

    int *Result = (int *) calloc(size * 2, sizeof(int));
    int *Follower = Result;

    while ((counter1 < size) && (counter2 < size))
    {
        if ((Pointer1[*(index1 + counter1)].reliab <= Pointer1[*(index2 + counter2)].reliab))
        {
            *Follower = *(index1 + counter1);
            Follower++;
            counter1++;
        }
        else
        {
            *Follower = *(index2 + counter2);
            Follower++;
            counter2++;
        }
    }//while

    if (counter1 == size)
    {
        memcpy(Follower, (index2 + counter2), sizeof(int)*(size-counter2));
    }
    else
    {
        memcpy(Follower, (index1 + counter1), sizeof(int)*(size-counter1));
    }

    Follower = Result;
    index1 = TemporalPointer;

    int i;
    for (i=0; i < 2 * size; i++)
    {
        *index1 = *Follower;
        index1++;
        Follower++;
    }

    free(Result);
}

//this is may be the fastest sort program;
//see the explination in quickSort function below
void  sort(EDGE *Pointer, int *index, int size)
{
    if (size == 2)
    {
        if ((Pointer[*index].reliab) > (Pointer[*(index+1)].reliab))
        {
            int Temp;
            Temp = *index;
            *index = *(index+1);
            *(index+1) = Temp;
        }
    }
    else if (size > 2)
    {
        sort(Pointer, index, size/2);
        sort(Pointer, (index + (size/2)), size/2);
        Mix(Pointer, index, (index + (size/2)), size/2);
    }
}

//this function tries to implement a nice idea explained below
//we need to sort edge array. Each edge element conisists of 16 bytes.
//In normal sort program we compare two elements in the array and exchange
//their place under some conditions to do the sorting. It is very probable
// that an edge element may change its place hundred of times which makes
//the sorting a very time consuming operation. The idea in this function
//is to give each edge element an index and move the index not the edge
//element. The edge need 4 bytes which makes the sorting operation faster.
// After finishingthe sorting of the indexes, we know the position of each index.
//So we know how to sort edges
void  quick_sort(EDGE *Pointer, int size)
{
    int *index = (int *) calloc(size, sizeof(int));
    int i;

    for (i=0; i<size; ++i)
        index[i] = i;

    sort(Pointer, index, size);

    EDGE * a = (EDGE *) calloc(size, sizeof(EDGE));
    for (i=0; i<size; ++i)
        a[i] = Pointer[*(index + i)];

    memcpy(Pointer, a, size*sizeof(EDGE));

    free(index);
    free(a);
}



void read_data(char *inputfile,float *Data, int length)
{
    printf("Reading the Wrapped Values form Binary File.............>");
    FILE *ifptr;
    ifptr = fopen(inputfile,"rb");
    if(ifptr == NULL) printf("Error opening the file\n");
    fread(Data,sizeof(float),length,ifptr);
    fclose(ifptr);
    printf(" Done.\n");
}

void write_data(char *outputfile,float *Data,int length)
{
    printf("Writing the Unwrapped Values to Binary File.............>");
    FILE *ifptr;
    ifptr = fopen(outputfile,"wb");
    if(ifptr == NULL) printf("Error opening the file\n");
    fwrite(Data,sizeof(float),length,ifptr);
    fclose(ifptr);
    printf(" Done.\n");
}

//---------------start quicker_sort algorithm --------------------------------
#define swap(x,y) {EDGE t; t=x; x=y; y=t;}
#define order(x,y) if (x.reliab > y.reliab) swap(x,y)
#define o2(x,y) order(x,y)
#define o3(x,y,z) o2(x,y); o2(x,z); o2(y,z)

typedef enum {yes, no} yes_no;

yes_no find_pivot(EDGE *left, EDGE *right, float *pivot_ptr)
{
    EDGE a, b, c, *p;

    a = *left;
    b = *(left + (right - left) /2 );
    c = *right;
    o3(a,b,c);

    if (a.reliab < b.reliab)
    {
        *pivot_ptr = b.reliab;
        return yes;
    }

    if (b.reliab < c.reliab)
    {
        *pivot_ptr = c.reliab;
        return yes;
    }

    for (p = left + 1; p <= right; ++p)
    {
        if (p->reliab != left->reliab)
        {
            *pivot_ptr = (p->reliab < left->reliab) ? left->reliab : p->reliab;
            return yes;
        }
        return no;
    }
}

EDGE *partition(EDGE *left, EDGE *right, float pivot)
{
    while (left <= right)
    {
        while (left->reliab < pivot)
            ++left;
        while (right->reliab >= pivot)
            --right;
        if (left < right)
        {
            swap (*left, *right);
            ++left;
            --right;
        }
    }
    return left;
}

void quicker_sort(EDGE *left, EDGE *right)
{
    EDGE *p;
    float pivot;

    if (find_pivot(left, right, &pivot) == yes)
    {
        p = partition(left, right, pivot);
        quicker_sort(left, p - 1);
        quicker_sort(p, right);
    }
}

//--------------end quicker_sort algorithm -----------------------------------

//--------------------start initialse pixels ----------------------------------
//initialse pixels. See the explination of the pixel class above.
//initially every pixel is a gorup by its self
void  initialisePIXELs(float *WrappedImage, PIXEL *pixel, int image_width, int image_height)
{
    PIXEL *pixel_pointer = pixel;
    float *wrapped_image_pointer = WrappedImage;
    int i, j;

    for (i=0; i < image_height; i++)
    {
        for (j=0; j < image_width; j++)
        {
            //pixel_pointer->x = j;
            //pixel_pointer->y = i;
            pixel_pointer->increment = 0;
            pixel_pointer->number_of_pixels_in_group = 1;
            pixel_pointer->value = *wrapped_image_pointer;
            pixel_pointer->reliability = 9999999+rand();
            pixel_pointer->head = pixel_pointer;
            pixel_pointer->last = pixel_pointer;
            pixel_pointer->next = NULL;
            pixel_pointer->new_group = 0;
            pixel_pointer->group = -1;
            pixel_pointer++;
            wrapped_image_pointer++;
        }
    }
}
//-------------------end initialise pixels -----------

//gamma function in the paper
float wrap(float pixel_value)
{
    float wrapped_pixel_value;
    if (pixel_value > PI)	wrapped_pixel_value = pixel_value - TWOPI;
    else if (pixel_value < -PI)	wrapped_pixel_value = pixel_value + TWOPI;
    else wrapped_pixel_value = pixel_value;
    return wrapped_pixel_value;
}

// pixelL_value is the left pixel,	pixelR_value is the right pixel
int find_wrap(float pixelL_value, float pixelR_value)
{
    float difference;
    int wrap_value;
    difference = pixelL_value - pixelR_value;

    if (difference > PI){
        //mexPrintf("Wrapping\n");
        wrap_value = -1;
    }
    else if (difference < -PI){
        //mexPrintf("Wrapping\n");
        wrap_value = 1;
    }
    else {
        wrap_value = 0;
    }

    return wrap_value;
}

void calculate_reliability(float *wrappedImage, PIXEL *pixel, int image_width, int image_height)
{
    int image_width_plus_one = image_width + 1;
    int image_width_minus_one = image_width - 1;
    PIXEL *pixel_pointer = pixel + image_width_plus_one;
    float *WIP = wrappedImage + image_width_plus_one; //WIP is the wrapped image pointer
    float H, V, D1, D2;
    int i, j;

    for (i = 1; i < image_height -1; ++i)
    {
        for (j = 1; j < image_width - 1; ++j)
        {
            H = wrap(*(WIP - 1) - *WIP) - wrap(*WIP - *(WIP + 1));
            V = wrap(*(WIP - image_width) - *WIP) - wrap(*WIP - *(WIP + image_width));
            D1 = wrap(*(WIP - image_width_plus_one) - *WIP) - wrap(*WIP - *(WIP + image_width_plus_one));
            D2 = wrap(*(WIP - image_width_minus_one) - *WIP) - wrap(*WIP - *(WIP + image_width_minus_one));
            pixel_pointer->reliability = H*H + V*V + D1*D1 + D2*D2;
            pixel_pointer++;
            WIP++;
        }
        pixel_pointer += 2;
        WIP += 2;
    }
}

//calculate the reliability of the horizental edges of the image
//it is calculated by adding the reliability of pixel and the relibility of
//its right neighbour
//edge is calculated between a pixel and its next neighbour
void  horizentalEDGEs(PIXEL *pixel, EDGE *edge, int image_width, int image_height)
{
    int i, j;
    EDGE *edge_pointer = edge;
    PIXEL *pixel_pointer = pixel;
    char mybuff1[50];

    for (i = 0; i < image_height; i++)
    {
        for (j = 0; j < image_width - 1; j++)
        {
            edge_pointer->pointer_1 = pixel_pointer;
            edge_pointer->pointer_2 = (pixel_pointer+1);
            edge_pointer->reliab = pixel_pointer->reliability + (pixel_pointer + 1)->reliability;
            edge_pointer->increment = find_wrap(pixel_pointer->value, (pixel_pointer + 1)->value);
            if(edge_pointer->increment != 0){
                //mexPrintf("Inc\n");
                sprintf (mybuff1, "%d", edge_pointer->increment);
                //mexPrintf(mybuff1);
                //mexPrintf("\n");
            }
            pixel_pointer++;
            edge_pointer++;
        }
        pixel_pointer++;
    }
}

//calculate the reliability of the vertical EDGEs of the image
//it is calculated by adding the reliability of pixel and the relibility of
//its lower neighbour in the image.
void  verticalEDGEs(PIXEL *pixel, EDGE *edge, int image_width, int image_height)
{
    int i, j;

    PIXEL *pixel_pointer = pixel;
    EDGE *edge_pointer = edge + (image_height) * (image_width - 1);
    char mybuff1[50];

    for (i=0; i<image_height - 1; i++)
    {
        for (j=0; j < image_width; j++)
        {
            edge_pointer->pointer_1 = pixel_pointer;
            edge_pointer->pointer_2 = (pixel_pointer + image_width);
            edge_pointer->reliab = pixel_pointer->reliability + (pixel_pointer + image_width)->reliability;
            edge_pointer->increment = find_wrap(pixel_pointer->value, (pixel_pointer + image_width)->value);
            if(edge_pointer->increment != 0){
                //mexPrintf("Inc\n");
                sprintf (mybuff1, "%d", edge_pointer->increment);
                //mexPrintf(mybuff1);
                //mexPrintf("\n");
            }
            pixel_pointer++;
            edge_pointer++;
        } //j loop
    } // i loop
}

//gather the pixels of the image into groups
void  gatherPIXELs(EDGE *edge, int image_width, int image_height)
{
    int k;
    char mybuff1[50], mybuff2[50],mybuff3[50];

    //Number of rialiable edges (not at the borders of the image)
    int no_EDGEs = (image_width - 1) * (image_height) + (image_width) * (image_height - 1);
    PIXEL *PIXEL1;
    PIXEL *PIXEL2;

    PIXEL *group1;
    PIXEL *group2;
    EDGE *pointer_edge = edge;
    int incremento;

    for (k = 0; k < no_EDGEs; k++)
    {
        PIXEL1 = pointer_edge->pointer_1;
        PIXEL2 = pointer_edge->pointer_2;

        //PIXEL 1 and PIXEL 2 belong to different groups
        //initially each pixel is a group by it self and one pixel can construct a group
        //no else or else if to this if
        if (PIXEL2->head != PIXEL1->head)
        {

            /*sprintf (mybuff1, "%f", pointer_edge->reliab);
            sprintf (mybuff2, "%f", PIXEL1->value);
            sprintf (mybuff3, "%f", PIXEL2->value);
            mexPrintf("Pix A: ");
            mexPrintf(mybuff2);
            mexPrintf(" - Pix B: ");
            mexPrintf(mybuff3);
            mexPrintf(" - ");
            mexPrintf(mybuff1);
            mexPrintf(" - ");*/
            //PIXEL 2 is alone in its group
            //merge this pixel with PIXEL 1 group and find the number of 2 pi to add
            //to or subtract to unwrap it
            if ((PIXEL2->next == NULL) && (PIXEL2->head == PIXEL2))
            {
                //mexPrintf("New B\n");
                PIXEL1->head->last->next = PIXEL2;
                PIXEL1->head->last = PIXEL2;
                (PIXEL1->head->number_of_pixels_in_group)++;
                PIXEL2->head=PIXEL1->head;
                PIXEL2->increment = PIXEL1->increment-pointer_edge->increment;
            }

                //PIXEL 1 is alone in its group
                //merge this pixel with PIXEL 2 group and find the number of 2 pi to add
                //to or subtract to unwrap it
            else if ((PIXEL1->next == NULL) && (PIXEL1->head == PIXEL1))
            {
                //mexPrintf("New A\n");
                PIXEL2->head->last->next = PIXEL1;
                PIXEL2->head->last = PIXEL1;
                (PIXEL2->head->number_of_pixels_in_group)++;
                PIXEL1->head = PIXEL2->head;
                PIXEL1->increment = PIXEL2->increment+pointer_edge->increment;
            }

                //PIXEL 1 and PIXEL 2 both have groups
            else
            {
                group1 = PIXEL1->head;
                group2 = PIXEL2->head;
                //the no. of pixels in PIXEL 1 group is large than the no. of PIXELs
                //in PIXEL 2 group.   Merge PIXEL 2 group to PIXEL 1 group
                //and find the number of wraps between PIXEL 2 group and PIXEL 1 group
                //to unwrap PIXEL 2 group with respect to PIXEL 1 group.
                //the no. of wraps will be added to PIXEL 2 grop in the future
                if (group1->number_of_pixels_in_group > group2->number_of_pixels_in_group)
                {
                    //mexPrintf("Big A\n");
                    //merge PIXEL 2 with PIXEL 1 group
                    group1->last->next = group2;
                    group1->last = group2->last;
                    group1->number_of_pixels_in_group = group1->number_of_pixels_in_group + group2->number_of_pixels_in_group;
                    incremento = PIXEL1->increment-pointer_edge->increment - PIXEL2->increment;
                    //merge the other pixels in PIXEL 2 group to PIXEL 1 group
                    while (group2 != NULL)
                    {
                        group2->head = group1;
                        group2->increment += incremento;
                        group2 = group2->next;
                    }
                }

                    //the no. of PIXELs in PIXEL 2 group is large than the no. of PIXELs
                    //in PIXEL 1 group.   Merge PIXEL 1 group to PIXEL 2 group
                    //and find the number of wraps between PIXEL 2 group and PIXEL 1 group
                    //to unwrap PIXEL 1 group with respect to PIXEL 2 group.
                    //the no. of wraps will be added to PIXEL 1 grop in the future
                else
                {
                    //mexPrintf("Big B\n");
                    //merge PIXEL 1 with PIXEL 2 group
                    group2->last->next = group1;
                    group2->last = group1->last;
                    group2->number_of_pixels_in_group = group2->number_of_pixels_in_group + group1->number_of_pixels_in_group;
                    incremento = PIXEL2->increment + pointer_edge->increment - PIXEL1->increment;
                    //merge the other pixels in PIXEL 2 group to PIXEL 1 group
                    while (group1 != NULL)
                    {
                        group1->head = group2;
                        group1->increment += incremento;
                        group1 = group1->next;
                    } // while
                } // else
            } //else
        } else {
            //mexPrintf("Same group\n");
        };//if

        pointer_edge++;
    }
}

//unwrap the image
void  unwrapImage(PIXEL *pixel, int image_width, int image_height)
{
    int i;
    int image_size = image_width * image_height;
    PIXEL *pixel_pointer=pixel;

    for (i = 0; i < image_size; i++)
    {
        pixel_pointer->value += TWOPI * (float)(pixel_pointer->increment);
        pixel_pointer++;
    }
}

//the input to this unwrapper is an array that contains the wrapped phase map.
//copy the image on the buffer passed to this unwrapper to over write the unwrapped
//phase map on the buffer of the wrapped phase map.
void  returnImage(PIXEL *pixel, float *unwrappedImage, int image_width, int image_height)
{
    int i;
    int image_size = image_width * image_height;
    float *unwrappedImage_pointer = unwrappedImage;
    PIXEL *pixel_pointer = pixel;

    for (i=0; i < image_size; i++)
    {
        *unwrappedImage_pointer = pixel_pointer->value;
        pixel_pointer++;
        unwrappedImage_pointer++;
    }
}


//the main function of the unwrapper
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    //Declarations of getting two arrays from Matlab
    //1)input wrapped image  of type float and 2)mask of type unsigned char
    float *WrappedImage = (float *)mxGetData(prhs[0]);
    int image_width = mxGetM(prhs[0]);
    int image_height = mxGetN(prhs[0]);

    //declare a place to store the unwrapped image and return it to Matlab
    const mwSize *dims = mxGetDimensions(prhs[0]);
    plhs[0] = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    float *UnwrappedImage = (float *)mxGetPr(plhs[0]);

    int i, j;
    int image_size = image_height * image_width;
    int two_image_size = 2 * image_size;

    int No_of_Edges = (image_width)*(image_height-1) + (image_width-1)*(image_height);

    PIXEL *pixel = (PIXEL *) calloc(image_size, sizeof(PIXEL));
    EDGE *edge = (EDGE *) calloc(No_of_Edges, sizeof(EDGE));;

    //initialise the pixels
    mexPrintf("Initialising pixels\n");
    initialisePIXELs(WrappedImage, pixel, image_width, image_height);

    mexPrintf("Calculating reliabililty\n");

    calculate_reliability(WrappedImage, pixel, image_width, image_height); //

    PIXEL *pixel_pointer = pixel;
    char mybuff1[50],mybuff2[50],mybuff3[50],mybuff4[50];

    /*for(i=0; i<image_size;++i){
        sprintf (mybuff1, "%f", pixel_pointer->reliability);
        mexPrintf(mybuff1);
        mexPrintf("\n");
        pixel_pointer++;
    }*/

    mexPrintf("Gathering edges\n");

    horizentalEDGEs(pixel, edge, image_width, image_height);
    verticalEDGEs(pixel, edge, image_width, image_height);

    mexPrintf("Sorting edges\n");

    //sort the EDGEs depending on their reiability. The PIXELs with higher relibility (small value) first
    //if your code stuck because of the quicker_sort() function, then use the quick_sort() function
    //run only one of the two functions (quick_sort() or quicker_sort() )
    //quick_sort(edge, No_of_Edges);
    //quicker_sort(edge, edge + No_of_Edges - 1);

    thrust::stable_sort(edge,edge+No_of_Edges-1,thrust::less<EDGE>());

    EDGE *edge_pointer = edge;
    PIXEL *PIXEL1;
    PIXEL *PIXEL2;
    double diff;

    int a;

    /*for(a=0; a<No_of_Edges;a++){
        PIXEL1 = edge_pointer->pointer_1;
		PIXEL2 = edge_pointer->pointer_2;
        diff = PIXEL1->value - PIXEL2->value;

        sprintf (mybuff1, "%d", edge_pointer->increment);
        sprintf (mybuff2, "%f", edge_pointer->reliab);
        sprintf (mybuff3, "%f", diff);

        mexPrintf(mybuff1);
        mexPrintf(" : ");
        mexPrintf(mybuff2);
        mexPrintf("---");
        mexPrintf(mybuff3);
        mexPrintf("\n");
        edge_pointer++;
    }*/

    mexPrintf("Gathering the pixels...\n");

    //gather PIXELs into groups
    gatherPIXELs(edge, image_width, image_height);

    pixel_pointer = pixel;
    /*for(i=0; i<image_size;++i){
        sprintf (mybuff1, "%d", pixel_pointer->increment);
        mexPrintf(mybuff1);
        mexPrintf("\n");
        pixel_pointer++;
    }*/

    //unwrap the whole image
    unwrapImage(pixel, image_width, image_height);

    //copy the image from PIXEL structure to the wrapped phase array passed to this function
    returnImage(pixel, UnwrappedImage, image_width, image_height);

    free(edge);
    free(pixel);
    return;
}