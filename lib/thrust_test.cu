#include "hip/hip_runtime.h"
//This program is written by Munther Gdeisat and Miguel Arevallilo Herra�ez to program the two-dimensional unwrapper
//entitled "Fast two-dimensional phase-unwrapping algorithm based on sorting by 
//reliability following a noncontinuous path"
//by  Miguel Arevallilo Herra�ez, David R. Burton, Michael J. Lalor, and Munther A. Gdeisat
//published in the Applied Optics, Vol. 41, No. 35, pp. 7437, 2002.
//This program is written on 15th August 2007
//The wrapped phase map is floating point data type. Also, the unwrapped phase map is foloating point
//#include <sys/malloc.h>
#include<stdio.h>
#include <stdlib.h>
#include <string.h>
#include "mex.h"   //--This one is required

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
//#include <stl.h>

__device__ static float PI = 3.141592654;
__device__ static float TWOPI = 6.283185307;

bool DEBUG = true;

struct saxpy_functor
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
    float operator()(const float& x, const float& y) const {
        return a * x + y;
    }
};

void saxpy_fast(float A, thrust::device_ptr<float> X, thrust::device_ptr<float> Y, int N)
{
    // Y <- A * X + Y
    thrust::transform(X, X+N, Y, Y+N, saxpy_functor(A));
}

__global__ void gpu_derivative_vertical(float* WrappedImage, float* DerivativeImage, int width, int height){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float* WIP;
    float* DIP;

    for(int i=index; i<width*height; i+=stride){
        WIP = WrappedImage + i;
        DIP = DerivativeImage + i;

        if(i/height == 0){
            *DIP = *(WIP+width) - *WIP;
        } else if(i + width > width*height){
            *DIP = *WIP - *(WIP-width);
        } else {
            *DIP = (*(WIP+width)-*WIP+*WIP-*(WIP-width))/2;
        }
    }
}

__global__ void gpu_derivative_horizontal(float* WrappedImage, float* DerivativeImage, int width, int height){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float* WIP;
    float* DIP;

    for(int i=index; i<width*height; i+=stride){
        WIP = WrappedImage + i;
        DIP = DerivativeImage + i;

        if(i % width == 0){
            *DIP = *(WIP+1) - *WIP;
        } else if (i % width == width-1){
            *DIP = *WIP - *(WIP-1);
        } else {
            *DIP = (*(WIP+1)-*WIP+*WIP-*(WIP-1))/2;
        }
    }
}

__global__ void gpu_unwrap(float* WrappedImage, float* UnwrappedImage, int width, int height){

}

//the main function of the unwrapper
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    //Declarations of getting two arrays from Matlab
    //1)input wrapped image  of type float and 2)mask of type unsigned char
    float *WrappedImage = (float *)mxGetData(prhs[0]);
    int image_width = mxGetM(prhs[0]);
    int image_height = mxGetN(prhs[0]);

    //declare a place to store the unwrapped image and return it to Matlab
    const mwSize *dims = mxGetDimensions(prhs[0]);
    plhs[0] = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    float *UnwrappedImage = (float *)mxGetPr(plhs[0]);

    int image_size = image_height * image_width;
    int two_image_size = 2 * image_size;

    int No_of_Edges = (image_width)*(image_height-1) + (image_width-1)*(image_height);

    int blockSize = 256; //1024;
    int numBlocks = 32; //(image_size + blockSize - 1) / blockSize;

    const int N = 6;
    int    keys[N] = {  1,   4,   2,   8,   5,   7};
    char values[N] = {'a', 'b', 'c', 'd', 'e', 'f'};

    thrust::sort_by_key(keys, keys + N, values);

    return;
}
